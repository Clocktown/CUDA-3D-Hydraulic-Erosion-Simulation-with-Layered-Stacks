#include "hip/hip_runtime.h"
#include "simulation.hpp"

namespace geo
{
namespace device
{

__constant__ Simulation simulation{};
__constant__ int2 offsets[4]{ { 1, 0 }, { 0, 1 }, { -1, 0 }, { 0, -1 } };

void setSimulation(const Simulation& data)
{
	CU_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(simulation), &data, sizeof(Simulation)));
}

}
}

#include "hip/hip_runtime.h"
#include "simulation.hpp"

namespace geo
{
namespace device
{

__constant__ Simulation simulation;
__constant__ int2 offsets[4]{ { 1, 0 }, { 0, 1 }, { -1, 0 }, { 0, -1 } };

void setSimulation(const Simulation& simulation)
{
	CU_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(device::simulation), &simulation, sizeof(Simulation)));
}

}
}

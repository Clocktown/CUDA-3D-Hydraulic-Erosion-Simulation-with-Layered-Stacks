#include "hip/hip_runtime.h"
#include "simulation.hpp"

namespace initNoise {
#include "cuda_noise.cuh"
}

namespace geo
{
namespace device
{

__device__ float fbm(glm::vec2 st, int octaves, float scale, int seed) {
	// Initial values
	float value = 0.0;
	float amplitude = .5;
	float frequency = 0.;
	//
	// Loop of octaves
	for (int i = 0; i < octaves; i++) {
		value += amplitude * initNoise::cudaNoise::simplexNoise(make_float3(st.x, st.y, 0), scale, seed);
		scale *= 2.;
		amplitude *= .5;
	}
	return value;
}

__global__ void initKernel()
{
	const glm::ivec2 index{ getLaunchIndex() };

	if (isOutside(index, simulation.gridSize))
	{
		return;
	}

	int flatIndex{ flattenIndex(index, simulation.gridSize) };


	float bedrockHeight = -30.f;
	float ceiling = FLT_MAX;
	simulation.layerCounts[flatIndex] = 1;

	/*if (index.x > (64.f / 256.f) * simulation.gridSize.x) {
		bedrockHeight = 50.f;
	}*/

	/*if (index.x <= (64.f / 256.f) * simulation.gridSize.x) {
		bedrockHeight = 0.f;
	}

	if (index.x > (200.f/256.f) * simulation.gridSize.x && index.y > (120.f/256.f) * simulation.gridSize.y && index.y < (136.f/256.f) * simulation.gridSize.y && index.x < (240.f/256.f) * simulation.gridSize.x) {
		bedrockHeight -= 25.f * (1.f - ((index.x - (200.f/256.f) * simulation.gridSize.x) / ((40.f/256.f) * simulation.gridSize.x) ));
	}

	if ((index.x <= (200.f / 256.f) * simulation.gridSize.x && index.y > (126.f / 256.f) * simulation.gridSize.y && index.y < (130.f / 256.f) * simulation.gridSize.y && index.x >(64.f / 256.f) * simulation.gridSize.x)) {
		bedrockHeight = 25.f * (((index.x - (64.f/256.f) * simulation.gridSize.x) / (simulation.gridSize.x * (200.f - 64.f) / 256.f)));
		ceiling = bedrockHeight + 3.f;
		simulation.layerCounts[flatIndex] = 2;

		simulation.heights[flatIndex + simulation.layerStride] = float4{ 50.f, 0.0f, 0.0f, FLT_MAX};
		simulation.sediments[flatIndex + simulation.layerStride] = 0.0f;
		simulation.fluxes[flatIndex + simulation.layerStride] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
		simulation.damages[flatIndex + simulation.layerStride] = 0.0f;
	}*/

	/*if (index.x >(64.f / 256.f) * simulation.gridSize.x && index.y >(120.f / 256.f) * simulation.gridSize.y && index.y < (136.f / 256.f) * simulation.gridSize.y && index.x < (240.f / 256.f) * simulation.gridSize.x) {
		bedrockHeight = 25.f;
		bedrockHeight = 30.f;
	}

	if (index.x >(64.f / 256.f) * simulation.gridSize.x && index.y >(200.f / 256.f) * simulation.gridSize.y && index.y < (216.f / 256.f) * simulation.gridSize.y && index.x < (240.f / 256.f) * simulation.gridSize.x) {
		bedrockHeight = 40.f;
	}

	if (index.x > (64.f/256.f) * simulation.gridSize.x && index.y > (40.f/256.f) * simulation.gridSize.y && index.y < (56.f/256.f) * simulation.gridSize.y && index.x < (240.f/256.f) * simulation.gridSize.x) {
		bedrockHeight = 40.f;
	}

	if (index.x > (120.f/256.f) * simulation.gridSize.x && index.y > (16.f/256.f) * simulation.gridSize.y && index.y < (240.f/256.f) * simulation.gridSize.y && index.x < (136.f/256.f) * simulation.gridSize.x) {
		bedrockHeight = 30.f;
	}

	if (index.y > (117.f/256.f) * simulation.gridSize.y && index.y < (139.f/256.f) * simulation.gridSize.y && index.x > (117.f/256.f) * simulation.gridSize.x  && index.x < (139.f/256.f) * simulation.gridSize.x) {
		bedrockHeight = 50.f;
	}*/

	//simulation.layerCounts[flatIndex] = 1;

	float scale = simulation.gridSize.x / 256.f;

	if ((index.x > 10 * scale) && (index.x < 246 * scale)) {
		if (index.y > 100 * scale && index.y < 110 * scale) {
			bedrockHeight = 20.f;
			if (index.x > 20 * scale && index.x < 236 * scale) {
				simulation.layerCounts[flatIndex] = 2;

				simulation.heights[flatIndex + simulation.layerStride] = float4{ bedrockHeight , 0.0f, 0.0f, FLT_MAX };
				simulation.sediments[flatIndex + simulation.layerStride] = 0.0f;
				simulation.fluxes[flatIndex + simulation.layerStride] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
				simulation.damages[flatIndex + simulation.layerStride] = 0.0f;

				bedrockHeight = -30.f;
				ceiling = bedrockHeight + 40.f;
			}
		}
	}

	if ((index.x > 10 * scale) && (index.x < 246 * scale)) {
		if (index.y > 50 * scale && index.y < 60 * scale) {
			bedrockHeight = 30.f;
			if (index.x > 20 * scale && index.x < 236 * scale) {
				simulation.layerCounts[flatIndex] = 2;

				simulation.heights[flatIndex + simulation.layerStride] = float4{ bedrockHeight , 0.0f, 0.0f, FLT_MAX};
				simulation.sediments[flatIndex + simulation.layerStride] = 0.0f;
				simulation.fluxes[flatIndex + simulation.layerStride] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
				simulation.damages[flatIndex + simulation.layerStride] = 0.0f;

				bedrockHeight = -30.f;
				ceiling = bedrockHeight + 50.f + 8.f * pow(1.f - fabsf(index.x - 128.f * scale) / (108.f * scale), 0.25f);
			}
		}
	}

	if ((index.x > 10 * scale) && (index.x < 246 * scale)) {
		if (index.y > 150 * scale && index.y < 160 * scale) {
			bedrockHeight = 10.f;
			if (index.x > 20 * scale && index.x < 236 * scale) {
				simulation.layerCounts[flatIndex] = 2;

				simulation.heights[flatIndex + simulation.layerStride] = float4{ bedrockHeight , 0.0f, 0.0f, FLT_MAX};
				simulation.sediments[flatIndex + simulation.layerStride] = 0.0f;
				simulation.fluxes[flatIndex + simulation.layerStride] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
				simulation.damages[flatIndex + simulation.layerStride] = 0.0f;

				bedrockHeight = -30.f;
				ceiling = bedrockHeight + 30.f + 8.f * pow(fabsf(index.x - 128.f * scale) / (108.f * scale), 4.f);
			}
		}
	}

	simulation.heights[flatIndex] = float4{ bedrockHeight, 0.0f, 0.0f, ceiling};
	simulation.sediments[flatIndex] = 0.0f;
	simulation.fluxes[flatIndex] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
	simulation.damages[flatIndex] = 0.0f;
	

	/*float bedrockHeight = 0.f;
	if (index.x > (64.f/256.f) * simulation.gridSize.x) {
		bedrockHeight = 50.f;
	}

	if (index.x > (70.f/256.f) * simulation.gridSize.x && index.y > (120.f/256.f) * simulation.gridSize.y && index.y < (136.f/256.f) * simulation.gridSize.y && index.x < (240.f/256.f) * simulation.gridSize.x) {
		bedrockHeight = 25.f * (256.f / simulation.gridSize.x) * (index.x - 70) / (240.f - 70.f);
	}

	simulation.layerCounts[flatIndex] = 1;
	simulation.heights[flatIndex] = float4{ bedrockHeight, 0.0f, 0.0f, FLT_MAX};
	simulation.sediments[flatIndex] = 0.0f;
	simulation.fluxes[flatIndex] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
	simulation.damages[flatIndex] = 0.0f;
	*/

	/*const float noiseVal1 = 5.f * fbm(index, 8, 0.01f * simulation.gridScale, 42);
	const float noiseVal2 = 10.f * (1.f + fbm(index, 8, 0.005f * simulation.gridScale, 69));

	if (float(index.x) / simulation.gridSize.x  > 0.25f)
	{
		simulation.layerCounts[flatIndex] = 2;
		simulation.heights[flatIndex] = float4{ noiseVal1 + 16.f * (simulation.gridSize.x - index.x) / simulation.gridSize.x, 0.0f, 0.0f, 30.0f};
		simulation.sediments[flatIndex] = 0.0f;
		simulation.fluxes[flatIndex] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
		simulation.damages[flatIndex] = 0.0f;

		simulation.heights[flatIndex + simulation.layerStride] = float4{ noiseVal2 + 30.0f + (16.f * index.x) / simulation.gridSize.x, 0.0f, glm::max(2.0f - noiseVal2, 0.f), FLT_MAX };
		simulation.sediments[flatIndex + simulation.layerStride] = 0.0f;
		simulation.fluxes[flatIndex + simulation.layerStride] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
		simulation.damages[flatIndex + simulation.layerStride] = 0.0f;
	}
	else
	{
		simulation.layerCounts[flatIndex] = 1;
		simulation.heights[flatIndex] = float4{ noiseVal1 + 16.f * (simulation.gridSize.x - index.x) / simulation.gridSize.x, 0.0f, 0.0f, FLT_MAX };
		simulation.sediments[flatIndex] = 0.0f;
		simulation.fluxes[flatIndex] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
		simulation.damages[flatIndex] = 0.0f;
	}*/

	/*const float noiseVal1 = 5.f * fbm(index, 8, 0.01f * simulation.gridScale, 42);
	const float noiseVal2 = 10.f * (1.f + fbm(index, 8, 0.05f * simulation.gridScale, 69));


	simulation.layerCounts[flatIndex] = 1;
	simulation.heights[flatIndex] = float4{ noiseVal2 + 16.f * (simulation.gridSize.x - index.x) / simulation.gridSize.x, 1.f, 0.0f, FLT_MAX};
	simulation.sediments[flatIndex] = 0.0f;
	simulation.fluxes[flatIndex] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
	simulation.damages[flatIndex] = 0.0f;*/


	// generate pre-made arches to test support check
	// scene with a lot of sand to demonstrate fake "regolith"

	/*const float noiseVal1 = 5.f * fbm(index, 8, 0.01f * simulation.gridScale, 42);
	const float noiseVal2 = 40.f * (1.f + fbm(index, 8, 0.005f * simulation.gridScale, 69));


	simulation.layerCounts[flatIndex] = 1;
	simulation.heights[flatIndex] = float4{ noiseVal2, 20.f, 0.0f, FLT_MAX};
	simulation.sediments[flatIndex] = 0.0f;
	simulation.fluxes[flatIndex] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
	simulation.damages[flatIndex] = 0.0f;*/

}

void init(const Launch& launch)
{
	CU_CHECK_KERNEL(initKernel<<<launch.gridSize, launch.blockSize>>>());
}

}
}

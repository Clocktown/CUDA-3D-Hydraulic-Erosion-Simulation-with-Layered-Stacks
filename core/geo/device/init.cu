#include "hip/hip_runtime.h"
#include "simulation.hpp"

namespace geo
{
namespace device
{

__global__ void initKernel()
{
	const glm::ivec2 index{ getLaunchIndex() };

	if (isOutside(index, simulation.gridSize))
	{
		return;
	}

	const int flatIndex{ flattenIndex(index, simulation.gridSize) };

	if (index.x > 64)
	{
		simulation.layerCounts[flatIndex] = 2;
		simulation.heights[flatIndex] = float4{ (simulation.gridSize.x - index.x) / 16.0f, 0.0f, 0.0f, 30.0f };
		simulation.sediments[flatIndex] = 0.0f;
		simulation.fluxes[flatIndex] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };

		simulation.heights[flatIndex + simulation.layerStride] = float4{ 30.0f + index.x / 16.0f, 2.0f, 2.0f, FLT_MAX };
		simulation.sediments[flatIndex + simulation.layerStride] = 0.0f;
		simulation.fluxes[flatIndex + simulation.layerStride] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
	}
	else
	{
		simulation.layerCounts[flatIndex] = 1;
		simulation.heights[flatIndex] = float4{ (simulation.gridSize.x - index.x) / 16.0f, 0.0f, 0.0f, FLT_MAX };
		simulation.sediments[flatIndex] = 0.0f;
		simulation.fluxes[flatIndex] = float4{ 0.0f, 0.0f, 0.0f, 0.0f };
	}
}

void init(const Launch& launch)
{
	CU_CHECK_KERNEL(initKernel<<<launch.gridSize, launch.blockSize>>>());
}

}
}

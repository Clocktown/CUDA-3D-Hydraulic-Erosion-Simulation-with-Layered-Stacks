#include "hip/hip_runtime.h"
#include "simulation.hpp"

namespace geo
{
namespace device
{

__global__ void pipeKernel()
{
	const glm::ivec2 index{ getLaunchIndex() };

	if (index.x >= simulation.gridSize.x || index.y >= simulation.gridSize.y)
	{
		return;
	}

	const float gridScale2{ 2.0f * simulation.gridScale };

	int flatIndex{ flattenIndex(index, simulation.gridSize) };
	const int layerCount{ simulation.layerCounts[flatIndex] };

	for (int layer{ 0 }; layer < layerCount; ++layer, flatIndex += simulation.layerStride)
	{
		const glm::vec4 height{ glm::cuda_cast(simulation.heights[flatIndex]) };
		const float sand{ height[BEDROCK] + height[SAND] };
		const float water{ sand + height[WATER] };

		glm::vec<4, char> pipe{ -1 };
		glm::vec4 heights{ sand };
		glm::vec4 flux{ glm::cuda_cast(simulation.fluxes[flatIndex]) };

		struct
		{
			glm::ivec2 index;
			int flatIndex;
			int layerCount;
			int layer;
			glm::vec4 height;
			float sand;
			float water;
		} neighbor;

		for (int i{ 0 }; i < 4; ++i)
		{
			neighbor.index = index + glm::cuda_cast(offsets[i]);

			if (isOutside(neighbor.index, simulation.gridSize))
			{
				continue;
			}

			neighbor.flatIndex = flattenIndex(neighbor.index, simulation.gridSize);
			neighbor.layerCount = simulation.layerCounts[neighbor.flatIndex];

			for (neighbor.layer = 0; neighbor.layer < neighbor.layerCount; ++neighbor.layer, neighbor.flatIndex += simulation.layerStride)
			{
				neighbor.height = glm::cuda_cast(simulation.heights[neighbor.flatIndex]);
				neighbor.sand = neighbor.height[BEDROCK] + neighbor.height[SAND];
				neighbor.water = neighbor.sand + neighbor.height[WATER];

				if (sand < neighbor.height[CEILING])
				{
					const float deltaHeight{ water - neighbor.water };
					const float crossSectionalArea{ simulation.gridScale * simulation.gridScale }; // dynamic?

					pipe[i] = static_cast<char>(neighbor.layer);
					heights[i] = neighbor.sand;
					flux[i] = (deltaHeight > 0.0f) *
						      glm::max(flux[i] - simulation.deltaTime * crossSectionalArea * simulation.gravity * deltaHeight * simulation.rGridScale, 0.0f);

					if (neighbor.height[CEILING] < FLT_MAX)
					{
						const float freeSpace{ (neighbor.height[CEILING] - neighbor.water) * simulation.gridScale * simulation.gridScale };
						const float takenSpace{ flux[i] * simulation.deltaTime };

						flux[i] *= glm::min(freeSpace / (takenSpace + glm::epsilon<float>()), 1.0f);
					}

					break;
				}
				else if (water < neighbor.height[CEILING])
				{
					break;
				}
			}
		}

		const glm::vec3 tangents[2]{ glm::normalize(glm::vec3{ gridScale2, heights[RIGHT] - heights[LEFT], 0.0f }),
									 glm::normalize(glm::vec3{ 0.0f, heights[UP] - heights[DOWN], gridScale2 }) };

		const glm::vec3 normal{ glm::cross(tangents[0], tangents[1]) };
		const float slope{ glm::sqrt(1.0f - normal.y * normal.y) }; // sin(alpha)

		const float totalFlux{ flux.x + flux.y + flux.z + flux.w };

		flux *= glm::min(height[WATER] * simulation.gridScale * simulation.gridScale /
					     (totalFlux * simulation.deltaTime + glm::epsilon<float>()), 1.0f);

		simulation.pipes[flatIndex] = glm::cuda_cast(pipe);
		simulation.slopes[flatIndex] = slope;
		simulation.fluxes[flatIndex] = glm::cuda_cast(flux);
	}
}

__global__ void transportKernel()
{
	const glm::ivec2 index{ getLaunchIndex() };

	if (index.x >= simulation.gridSize.x || index.y >= simulation.gridSize.y)
	{
		return;
	}

	int flatIndex{ flattenIndex(index, simulation.gridSize) };
	const int layerCount{ simulation.layerCounts[flatIndex] };

	const float integrationScale{ simulation.rGridScale * simulation.rGridScale * simulation.deltaTime };

	for (int layer{ 0 }; layer < layerCount; ++layer, flatIndex += simulation.layerStride)
	{
		glm::vec4 height{ glm::cuda_cast(simulation.heights[flatIndex]) };
		float sediment{ simulation.sediments[flatIndex] };
		glm::vec4 flux{ glm::cuda_cast(simulation.fluxes[flatIndex]) };
		glm::vec4 sedimentFlux{ sediment * flux };

		struct
		{
			glm::ivec2 index;
			int flatIndex;
			int flatIndex4;
			int layerCount;
			int layer;
			float sediment;
			float flux;
		} neighbor;

		for (int i{ 0 }; i < 4; ++i)
		{
			neighbor.index = index + glm::cuda_cast(offsets[i]);

			if (isOutside(neighbor.index, simulation.gridSize))
			{
				continue;
			}

			neighbor.flatIndex = flattenIndex(neighbor.index, simulation.gridSize);
			neighbor.layerCount = simulation.layerCounts[neighbor.flatIndex];
			
			const int direction{ (i + 2) % 4 };

			for (neighbor.layer = 0; neighbor.layer < neighbor.layerCount; ++neighbor.layer, neighbor.flatIndex += simulation.layerStride)
			{
				neighbor.flatIndex4 = 4 * neighbor.flatIndex;

				if (reinterpret_cast<char*>(simulation.pipes)[neighbor.flatIndex4 + direction] == layer)
				{
					neighbor.sediment = simulation.sediments[neighbor.flatIndex];
					neighbor.flux = reinterpret_cast<float*>(simulation.fluxes)[neighbor.flatIndex4 + direction];

					flux[i] -= neighbor.flux;
					sedimentFlux[i] -= neighbor.sediment * neighbor.flux;
				}
			}
		}

		float avgWater = height[WATER];
		height[WATER] = glm::clamp(height[WATER] - integrationScale * (flux.x + flux.y + flux.z + flux.w), 0.0f, height[CEILING] - height[BEDROCK] - height[SAND]);
		avgWater = 0.5f * (avgWater + height[WATER]);
		
		sediment = glm::max(sediment - integrationScale * (sedimentFlux.x + sedimentFlux.y + sedimentFlux.z + sedimentFlux.w), 0.0f);

		const glm::vec2 velocity{ glm::vec2(flux[RIGHT] - flux[LEFT], flux[UP] - flux[DOWN]) / (avgWater * simulation.gridScale + glm::epsilon<float>()) };
		const float terrainSlope{ glm::max(simulation.slopes[flatIndex], simulation.minTerrainSlope) };
		const float sedimentCapacity{ simulation.sedimentCapacityConstant * terrainSlope * glm::length(velocity) };

		if (sedimentCapacity > sediment)
		{
			const float deltaSand{ glm::min(simulation.dissolvingConstant * (sedimentCapacity - sediment), height[SAND]) };
			height[SAND] -= deltaSand;
			sediment += deltaSand;
		}
		else
		{
			const float deltaSediment{ glm::min(simulation.depositionConstant * (sediment - sedimentCapacity),  height[CEILING] - height[BEDROCK] - height[WATER]) };
			sediment -= deltaSediment;
			height[SAND] += deltaSediment;
		}

		simulation.heights[flatIndex] = glm::cuda_cast(height);
		simulation.sediments[flatIndex] = sediment;
	}
}

void transport(const Launch& launch)
{
	CU_CHECK_KERNEL(pipeKernel<<<launch.gridSize, launch.blockSize>>>());
	CU_CHECK_KERNEL(transportKernel<<<launch.gridSize, launch.blockSize>>>());
}

}
}

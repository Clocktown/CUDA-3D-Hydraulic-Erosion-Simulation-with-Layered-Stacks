#include "hip/hip_runtime.h"
#include "simulation.hpp"

#include <thrust/execution_policy.h>
#include <thrust/sort.h>

namespace geo
{
namespace device
{

__global__ void fillIndicesKernel()
{
	const glm::ivec2 index{ getLaunchIndex() };

	if (index.x >= simulation.gridSize.x || index.y >= simulation.gridSize.y)
	{
		return;
	}

	int flatIndex{ flattenIndex(index, simulation.gridSize) };
	const int layerCount{ simulation.layerCounts[flatIndex] };

	for (int layer{ 0 }; layer < layerCount; ++layer, flatIndex += simulation.layerStride)
	{
		int idx = atomicAdd(simulation.atomicCounter, 1);
		simulation.indices[idx] = flatIndex;
	}
	
}

int fillIndices(const Launch& launch, int* atomicCounter, int* indices)
{
	int count = 0;
	hipMemcpy(atomicCounter, &count, sizeof(int), hipMemcpyHostToDevice);
	CU_CHECK_KERNEL(fillIndicesKernel<<<launch.gridSize, launch.blockSize>>>());
	hipMemcpy(&count, atomicCounter, sizeof(int), hipMemcpyDeviceToHost);
	return count;
}

}
}
